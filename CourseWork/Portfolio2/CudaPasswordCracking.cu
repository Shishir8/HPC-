#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>

// nvcc -o CudaPasswordCracking CudaPasswordCracking.cu

__device__ int passcrack(char *crack){

   char pass1[]="SH2973";
   char pass2[]="KR3097";
   char pass3[]="PK9736";
   char pass4[]="BM4397";

   char *s1 = crack;
   char *s2 = crack;
   char *s3 = crack;
   char *s4 = crack;

   char *p1 = pass1;
   char *p2 = pass2;
   char *p3 = pass3;
   char *p4 = pass4;

   while(*s1 == *p1){
    
       if(*s1 == '\0'){
          return 1;
       }
       s1++;
       p1++;
   }
   
    while(*s2 == *p2){
    
       if(*s2 == '\0'){
          return 1;
       }
       s2++;
       p2++;
   }

    while(*s3 == *p3){
    
       if(*s3 == '\0'){
          return 1;
       }
       s3++;
       p3++;
   }

    while(*s4 == *p4){
    
       if(*s4 == '\0'){
          return 1;
       }
       s4++;
       p4++;
   }
 return 0;
}

__global__ void kernel() {

 
  char alphabet[26] = {'A','B','C','D','E','F','G','H','I','J','K','L','M','N','O','P','Q','R','S','T','U','V','W','X','Y','Z'};
  

  char num[10] = {'0','1','2','3','4','5','6','7','8','9'};
  

  char crack[7];
  crack[6] = '\0';
  int s, h, k, r;

     for(s=0;s<10;s++){
      for(h=0; h<10; h++){
       for(k=0; k<10; k++){
        for(r=0; r<10; r++){

        crack[0] = alphabet[blockIdx.x];
        crack[1] = alphabet[threadIdx.x];
        crack[2] = num[s];
        crack[3] = num[h];
        crack[4] = num[k];
        crack[5] = num[r];

        if(passcrack(crack)){
            printf("Password successfully cracked: %s\n", crack);
        }

       }
      }
     }
    }
   }

int time_difference(struct timespec *start, struct timespec *finish, long long int *difference)
 {
     long long int ds =  finish->tv_sec - start->tv_sec;
     long long int dn =  finish->tv_nsec - start->tv_nsec;

     if(dn < 0 )
     {
      ds--;
      dn += 1000000000;
      }

     *difference = ds * 1000000000 + dn;
    return !(*difference > 0);
}


int main(int argc, char *argv[])
{

    struct timespec start, finish;  
    long long int time_elapsed;

    clock_gettime(CLOCK_MONOTONIC, &start);

    kernel <<<26, 26>>>();

    hipDeviceSynchronize();


    clock_gettime(CLOCK_MONOTONIC, &finish);
    time_difference(&start, &finish, &time_elapsed);
     printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed,
                                           (time_elapsed/1.0e9));
  return 0;
}







